#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>  
#include <fstream>
#include <sstream>    
#include <opencv2\core\core.hpp>
#include <opencv2\highgui\highgui.hpp>
#include <opencv2\opencv.hpp>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <ctime>
#include <wchar.h>
#include <thread>



#define NUM_OF_CONV_LAYERS 12
#define NUM_OF_PCA_DIMS 15

clock_t start_t, end_t;
using namespace std;
using namespace cv;
const char *inputpath, *outputpath, *modelpath = "model";
int scale_factor = 2;
int noise_level = 3;
vector<string> imageSet;


#define checkCUDNN(expression)                                  \
  {                                                             \
  hipdnnStatus_t status = (expression);                        \
if (status != HIPDNN_STATUS_SUCCESS) { \
	std::cerr << "Error on line " << __LINE__ << ": "       \
	<< hipdnnGetErrorString(status) << std::endl;            \
	std::exit(EXIT_FAILURE);                                \
}                                                           \
  }



__global__ void conv_bias_add(float *vector, const float* bias, const int sizeOfmaps, const int numOfmaps)
{
	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
	unsigned int idx = iy * sizeOfmaps + ix;
	vector[idx] += bias[iy];
}





static char* optarg = NULL;
static int optind = 1;
static char getopt(int argc, char* const argv[], const wchar_t* optstring)
{
	if (optind >= argc)
		return ' ';
	char opt = argv[optind][1];
	const wchar_t* p = wcschr(optstring, (wchar_t)opt);
	if (p == NULL)
		return L'?';
	optarg = NULL;

	if (p[1] == L':')
	{
		optind++;
		if (optind >= argc)
			return L'?';

		optarg = (char*)(argv[optind]);
	}

	optind++;
	return opt;
}


static void print_usage()
{
	fprintf(stderr, "Usage: sr-cuda -i infile -o outfile [options]...\n\n");
	fprintf(stderr, "  -h                   show this help\n");
	fprintf(stderr, "  -i input-path        input image path (jpg/png) or directory\n");
	fprintf(stderr, "  -o output-path       output image path (jpg/png) or directory\n");
	fprintf(stderr, "  -n noise-level       denoise level (-1/0/1/2/3/4/5/6/7/8/9/10, default=3)\n");
	fprintf(stderr, "  -s scale             upscale ratio (2/3/4, default=2)\n");
	fprintf(stderr, "  -m model-path        model path (default=model)\n");
}




void pixelShuffle(int rows, int cols, Mat& image_H, int item, float* outputBuffer)
{
	int image_size = rows*cols;
	int sf_pow = scale_factor * scale_factor;
	for (int row = 0; row < rows; row++)
	{
		for (int col = 0; col < cols; col++)
		{
			int ptr = row * cols + col;
			for (int i = 0; i < 3; i++)
			{
				for (int j = 0; j < scale_factor; j++)
				{
					for (int k = 0; k < scale_factor; k++)
					{
						float data = outputBuffer[(sf_pow * i + scale_factor * j + k)* image_size + ptr];
						image_H.at<Vec3b>(scale_factor * row + j, scale_factor * col + k)[2 - i] = (uchar)(data < 1.f ? data * 255.f : 255);
					}
				}
			}
		}
	}
	char temp[20];
	if (imageSet.size() == 1)
	{
		imwrite(string(outputpath), image_H);
	}
	else
	{
		sprintf(temp, (string("%0")+ to_string((int)log10(imageSet.size()) + 1)+"d").c_str(), item + 1);
		imwrite(outputpath + string("\\") + string((char*)temp) + ".png", image_H);
	}
}
int main(int argc, char** argv)
{
	if (argc == 1)
	{
		print_usage();
		return -1;
	}
	char opt;
	while ((opt = getopt(argc, argv, L"i:o:n:s:t:m:g:j:f:vxh")) != ' ')
	{
		switch (opt)
		{
		case 'i':
			inputpath = optarg;
			break;
		case 'o':
			outputpath = optarg;
			break;
		case 'n':
			noise_level = atoi(optarg);
			break;
		case 's':
			scale_factor = atoi(optarg);
			break;
		case 'm':
			modelpath = optarg;
			break;
		case 'h':
		default:
			print_usage();
			return -1;
		}
	}
	int input_channels;
	string modelpath_str;
	if (noise_level == -1)
	{
		input_channels = 18;
		modelpath_str = string(modelpath) + string("\\srnf_x") + to_string(scale_factor) + string(".acc");

	}
	else if (0 <= noise_level && noise_level <= 10)
	{
		input_channels = 19;
		modelpath_str = string(modelpath) + string("\\sr_x") + to_string(scale_factor) + string(".acc");
	}
	else
	{
		return -1;
	}
	modelpath = modelpath_str.c_str();
	std::ifstream model(modelpath, std::ios::in | ios::binary);
	model.read((char*)&scale_factor, 4);
	float pca_kernel[NUM_OF_PCA_DIMS];
	float* conv_kernel_weights[NUM_OF_CONV_LAYERS];
	float* conv_kernel_bias[NUM_OF_CONV_LAYERS];
	conv_kernel_weights[0] = new float[9 * input_channels * 128];
	conv_kernel_bias[0] = new float[128];
	for (int i = 1; i < NUM_OF_CONV_LAYERS - 1; i++)
	{
		conv_kernel_weights[i] = new float[9 * 128 * 128];
		conv_kernel_bias[i] = new float[128];
	}
	conv_kernel_weights[11] = new float[9 * 128 * 3 * scale_factor*scale_factor];
	conv_kernel_bias[11] = new float[3 * scale_factor*scale_factor];

	for (int i = 0; i < NUM_OF_PCA_DIMS; i++)
	{
		model.read((char*)&pca_kernel[i], 4);
	}
	for (int j = 0; j < 9 * input_channels * 128; j++)
	{
		model.read((char*)&conv_kernel_weights[0][j], 4);
	}
	for (int j = 0; j < 128; j++)
	{
		model.read((char*)&conv_kernel_bias[0][j], 4);
	}
	for (int i = 1; i < NUM_OF_CONV_LAYERS - 1; i++)
	{
		for (int j = 0; j < 9 * 128 * 128; j++)
		{
			model.read((char*)&conv_kernel_weights[i][j], 4);
		}
		for (int j = 0; j < 128; j++)
		{
			model.read((char*)&conv_kernel_bias[i][j], 4);
		}
	}

	for (int j = 0; j < 9 * 128 * 3 * scale_factor*scale_factor; j++)
	{
		model.read((char*)&conv_kernel_weights[11][j], 4);
	}
	for (int j = 0; j < 3 * scale_factor*scale_factor; j++)
	{
		model.read((char*)&conv_kernel_bias[11][j], 4);
	}
	model.close();
	hipdnnHandle_t handle;
	hipdnnCreate(&handle);

	Mat image;
	image = imread(inputpath);
	
	if (image.empty())
	{
		glob(inputpath, imageSet, false);
		if (imageSet.empty())
			return -1;
		else
			image = imread(imageSet[0]);
	}
	else
	{
		imageSet.push_back(inputpath);
	}
	Mat image_H(image.rows*scale_factor, image.cols*scale_factor, CV_8UC3);
	int image_size = image.rows*image.cols;
	int buffer_size = image_size * 3 * sizeof(float);
	float* inputBuffer = (float *)malloc(input_channels * image_size * sizeof(float));
	float* outputBuffer = (float *)malloc(buffer_size * scale_factor * scale_factor);


	// 输入张量的描述
	hipdnnTensorDescriptor_t input_descriptor_first;
	checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor_first));
	checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor_first,
		/*format=*/HIPDNN_TENSOR_NCHW,
		/*dataType=*/HIPDNN_DATA_FLOAT,
		/*batch_size=*/1,
		/*channels=*/input_channels,
		/*image_height=*/image.rows,
		/*image_width=*/image.cols));

	// 卷积核的描述（形状、格式）
	hipdnnFilterDescriptor_t kernel_descriptor_first;
	checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor_first));
	checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor_first,
		/*dataType=*/HIPDNN_DATA_FLOAT,
		/*format=*/HIPDNN_TENSOR_NCHW,	// 注意是 NCHW
		/*out_channels=*/128,
		/*in_channels=*/input_channels,
		/*kernel_height=*/3,
		/*kernel_width=*/3));

	// 卷积操作的描述（步长、填充等等）
	hipdnnConvolutionDescriptor_t convolution_descriptor_first;
	checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor_first));
	checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor_first,
		/*pad_height=*/1,
		/*pad_width=*/1,
		/*vertical_stride=*/1,
		/*horizontal_stride=*/1,
		/*dilation_height=*/1,
		/*dilation_width=*/1,
		/*mode=*/HIPDNN_CROSS_CORRELATION, // HIPDNN_CONVOLUTION
		/*computeType=*/HIPDNN_DATA_FLOAT));

	// 卷积输出张量的描述
	hipdnnTensorDescriptor_t output_descriptor_first;
	checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor_first));
	checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor_first,
		/*format=*/HIPDNN_TENSOR_NCHW,
		/*dataType=*/HIPDNN_DATA_FLOAT,
		/*batch_size=*/1,
		/*channels=*/128,
		/*image_height=*/image.rows,
		/*image_width=*/image.cols));

	// 卷积算法的描述
  // cudnn_tion_fwd_algo_gemm——将卷积建模为显式矩阵乘法，
  // cudnn_tion_fwd_algo_fft——它使用快速傅立叶变换(FFT)进行卷积或
  // cudnn_tion_fwd_algo_winograd——它使用Winograd算法执行卷积。
	hipdnnConvolutionFwdAlgo_t convolution_algorithm_first;
	checkCUDNN(
		hipdnnGetConvolutionForwardAlgorithm(handle,
			input_descriptor_first,
			kernel_descriptor_first,
			convolution_descriptor_first,
			output_descriptor_first,
			HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, // CUDNN_CONVOLUTION_FWD_SPECIFY_​WORKSPACE_LIMIT（在内存受限的情况下，memoryLimitInBytes 设置非 0 值）
			/*memoryLimitInBytes=*/0,
			&convolution_algorithm_first));



	hipdnnTensorDescriptor_t input_descriptor_mid;
	checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor_mid));
	checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor_mid,
		/*format=*/HIPDNN_TENSOR_NCHW,	// 注意是 NHWC，TensorFlow更喜欢以 NHWC 格式存储张量(通道是变化最频繁的地方，即 BGR)，而其他一些更喜欢将通道放在前面
		/*dataType=*/HIPDNN_DATA_FLOAT,
		/*batch_size=*/1,
		/*channels=*/128,
		/*image_height=*/image.rows,
		/*image_width=*/image.cols));

	// 卷积核的描述（形状、格式）
	hipdnnFilterDescriptor_t kernel_descriptor_mid;
	checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor_mid));
	checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor_mid,
		/*dataType=*/HIPDNN_DATA_FLOAT,
		/*format=*/HIPDNN_TENSOR_NCHW,	// 注意是 NCHW
		/*out_channels=*/128,
		/*in_channels=*/128,
		/*kernel_height=*/3,
		/*kernel_width=*/3));

	// 卷积操作的描述（步长、填充等等）
	hipdnnConvolutionDescriptor_t convolution_descriptor_mid;
	checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor_mid));
	checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor_mid,
		/*pad_height=*/1,
		/*pad_width=*/1,
		/*vertical_stride=*/1,
		/*horizontal_stride=*/1,
		/*dilation_height=*/1,
		/*dilation_width=*/1,
		/*mode=*/HIPDNN_CROSS_CORRELATION, // HIPDNN_CONVOLUTION
		/*computeType=*/HIPDNN_DATA_FLOAT));


	// 卷积输出张量的描述
	hipdnnTensorDescriptor_t output_descriptor_mid;
	checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor_mid));
	checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor_mid,
		/*format=*/HIPDNN_TENSOR_NCHW,
		/*dataType=*/HIPDNN_DATA_FLOAT,
		/*batch_size=*/1,
		/*channels=*/128,
		/*image_height=*/image.rows,
		/*image_width=*/image.cols));

	// 卷积算法的描述
  // cudnn_tion_fwd_algo_gemm——将卷积建模为显式矩阵乘法，
  // cudnn_tion_fwd_algo_fft——它使用快速傅立叶变换(FFT)进行卷积或
  // cudnn_tion_fwd_algo_winograd——它使用Winograd算法执行卷积。
	hipdnnConvolutionFwdAlgo_t convolution_algorithm_mid;
	checkCUDNN(
		hipdnnGetConvolutionForwardAlgorithm(handle,
			input_descriptor_mid,
			kernel_descriptor_mid,
			convolution_descriptor_mid,
			output_descriptor_mid,
			HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, // CUDNN_CONVOLUTION_FWD_SPECIFY_​WORKSPACE_LIMIT（在内存受限的情况下，memoryLimitInBytes 设置非 0 值）
			/*memoryLimitInBytes=*/0,
			&convolution_algorithm_mid));



	hipdnnTensorDescriptor_t input_descriptor_last;
	checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor_last));
	checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor_last,
		/*format=*/HIPDNN_TENSOR_NCHW,	// 注意是 NCHW，TensorFlow更喜欢以 NHWC 格式存储张量(通道是变化最频繁的地方，即 BGR)，而其他一些更喜欢将通道放在前面
		/*dataType=*/HIPDNN_DATA_FLOAT,
		/*batch_size=*/1,
		/*channels=*/128,
		/*image_height=*/image.rows,
		/*image_width=*/image.cols));

	// 卷积核的描述（形状、格式）
	hipdnnFilterDescriptor_t kernel_descriptor_last;
	checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor_last));
	checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor_last,
		/*dataType=*/HIPDNN_DATA_FLOAT,
		/*format=*/HIPDNN_TENSOR_NCHW,	// 注意是 NCHW
		/*out_channels=*/3 * scale_factor*scale_factor,
		/*in_channels=*/128,
		/*kernel_height=*/3,
		/*kernel_width=*/3));

	// 卷积操作的描述（步长、填充等等）
	hipdnnConvolutionDescriptor_t convolution_descriptor_last;
	checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor_last));
	checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor_last,
		/*pad_height=*/1,
		/*pad_width=*/1,
		/*vertical_stride=*/1,
		/*horizontal_stride=*/1,
		/*dilation_height=*/1,
		/*dilation_width=*/1,
		/*mode=*/HIPDNN_CROSS_CORRELATION, // HIPDNN_CONVOLUTION
		/*computeType=*/HIPDNN_DATA_FLOAT));

	// 卷积输出张量的描述
	hipdnnTensorDescriptor_t output_descriptor_last;
	checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor_last));
	checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor_last,
		/*format=*/HIPDNN_TENSOR_NCHW,
		/*dataType=*/HIPDNN_DATA_FLOAT,
		/*batch_size=*/1,
		/*channels=*/3 * scale_factor*scale_factor,
		/*image_height=*/image.rows,
		/*image_width=*/image.cols));

	// 卷积算法的描述
  // cudnn_tion_fwd_algo_gemm——将卷积建模为显式矩阵乘法，
  // cudnn_tion_fwd_algo_fft——它使用快速傅立叶变换(FFT)进行卷积或
  // cudnn_tion_fwd_algo_winograd——它使用Winograd算法执行卷积。
	hipdnnConvolutionFwdAlgo_t convolution_algorithm_last;
	checkCUDNN(
		hipdnnGetConvolutionForwardAlgorithm(handle,
			input_descriptor_last,
			kernel_descriptor_last,
			convolution_descriptor_last,
			output_descriptor_last,
			HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, // CUDNN_CONVOLUTION_FWD_SPECIFY_​WORKSPACE_LIMIT（在内存受限的情况下，memoryLimitInBytes 设置非 0 值）
			/*memoryLimitInBytes=*/0,
			&convolution_algorithm_last));

	// 计算 cuDNN 它的操作需要多少内存
	size_t workspace_bytes{ 0 };
	checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(handle,
		input_descriptor_mid,
		kernel_descriptor_mid,
		convolution_descriptor_mid,
		output_descriptor_mid,
		convolution_algorithm_mid,
		&workspace_bytes));

	hipdnnActivationDescriptor_t activation_descriptor;
	checkCUDNN(hipdnnCreateActivationDescriptor(&activation_descriptor));
	checkCUDNN(hipdnnSetActivationDescriptor(activation_descriptor,
		HIPDNN_ACTIVATION_RELU,
		HIPDNN_PROPAGATE_NAN,
		/*relu_coef=*/0));


	// *************************************************************************
		// 分配内存， 从 hipdnnGetConvolutionForwardWorkspaceSize 计算而得
	void* d_workspace{ nullptr };
	hipMalloc(&d_workspace, workspace_bytes);


	// 从 hipdnnGetConvolution2dForwardOutputDim 计算而得
	int size_of_featuremap = 128 * image_size * sizeof(float);

	float* d_featuremap_1{ nullptr };
	hipMalloc(&d_featuremap_1, size_of_featuremap);

	float* d_featuremap_2{ nullptr };
	hipMalloc(&d_featuremap_2, size_of_featuremap);
	float* d_kernel[NUM_OF_CONV_LAYERS];
	hipMalloc(&d_kernel[0], input_channels * 128 * 3 * 3 * sizeof(float));
	hipMemcpy(d_kernel[0], conv_kernel_weights[0], input_channels * 128 * 3 * 3 * sizeof(float), hipMemcpyHostToDevice);
	for (int i = 1; i < NUM_OF_CONV_LAYERS - 1; i++)
	{
		hipMalloc(&d_kernel[i], 128 * 128 * 3 * 3 * sizeof(float));
		hipMemcpy(d_kernel[i], conv_kernel_weights[i], 128 * 128 * 3 * 3 * sizeof(float), hipMemcpyHostToDevice);
	}
	hipMalloc(&d_kernel[NUM_OF_CONV_LAYERS - 1], 128 * 3 * scale_factor*scale_factor * 3 * 3 * sizeof(float));
	hipMemcpy(d_kernel[NUM_OF_CONV_LAYERS - 1], conv_kernel_weights[NUM_OF_CONV_LAYERS - 1], 128 * 3 * scale_factor*scale_factor * 3 * 3 * sizeof(float), hipMemcpyHostToDevice);

	float* d_bias[NUM_OF_CONV_LAYERS];
	for (int i = 0; i < NUM_OF_CONV_LAYERS - 1; i++)
	{
		hipMalloc(&d_bias[i], 128 * sizeof(float));
		hipMemcpy(d_bias[i], conv_kernel_bias[i], 128 * sizeof(float), hipMemcpyHostToDevice);
	}
	hipMalloc(&d_bias[NUM_OF_CONV_LAYERS - 1], 3 * scale_factor*scale_factor * sizeof(float));
	hipMemcpy(d_bias[NUM_OF_CONV_LAYERS - 1], conv_kernel_bias[NUM_OF_CONV_LAYERS - 1], 3 * scale_factor*scale_factor * sizeof(float), hipMemcpyHostToDevice);

	for (int i = 0; i < NUM_OF_PCA_DIMS; i++)
	{
		for (int row = 0; row < image.rows; row++)
		{
			for (int col = 0; col < image.cols; col++)
			{
				inputBuffer[(3 + i)* image_size + row * image.cols + col] = pca_kernel[i];
			}
		}
	}

	if (noise_level != -1)
	{

		for (int row = 0; row < image.rows; row++)
		{
			for (int col = 0; col < image.cols; col++)
			{
				inputBuffer[(input_channels - 1) * image_size + row * image.cols + col] = noise_level / 255.f;
			}
		}
	}

	for (int i = 0; i < NUM_OF_CONV_LAYERS; i++)
	{
		delete conv_kernel_weights[i];
		delete conv_kernel_bias[i];
	}
	thread pixelShuffle_thread;
	for (int item = 0; item < imageSet.size(); item++)
	{
		start_t = clock();
		image = imread(imageSet[item]);
		for (int row = 0; row < image.rows; row++)
		{
			for (int col = 0; col < image.cols; col++)
			{
				inputBuffer[row*image.cols + col] = image.at<Vec3b>(row, col)[2] / 255.f;
				inputBuffer[image_size + row * image.cols + col] = image.at<Vec3b>(row, col)[1] / 255.f;
				inputBuffer[2 * image_size + row * image.cols + col] = image.at<Vec3b>(row, col)[0] / 255.f;
			}
		}

		hipMemcpy(d_featuremap_1, inputBuffer, input_channels * image_size * sizeof(float), hipMemcpyHostToDevice);
		const float alpha = 1.0f, beta = 0.0f;
		// 真正的卷积操作 ！！！前向卷积
		checkCUDNN(hipdnnConvolutionForward(handle,
			&alpha,
			input_descriptor_first,
			d_featuremap_1,
			kernel_descriptor_first,
			d_kernel[0],
			convolution_descriptor_first,
			convolution_algorithm_first,
			d_workspace, // 注意，如果我们选择不需要额外内存的卷积算法，d_workspace可以为nullptr。
			workspace_bytes,
			&beta,
			output_descriptor_first,
			d_featuremap_2));

		int nx = image.rows*image.cols;
		int ny = 128;
		int dimx = 64;
		int dimy = 2;
		dim3 block(dimx, dimy);
		dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
		conv_bias_add << <grid, block >> > (d_featuremap_2, d_bias[0], nx, ny);



		// 前向 Relu 激活函数
		checkCUDNN(hipdnnActivationForward(handle,
			activation_descriptor,
			&alpha,
			output_descriptor_first,
			d_featuremap_2,
			&beta,
			output_descriptor_first,
			d_featuremap_2));
		//hipdnnDestroyActivationDescriptor(activation_descriptor);
		
		for (int i = 1; i < NUM_OF_CONV_LAYERS - 1; i++)
		{
			if (i % 2 == 0)
			{
				const float alpha = 1.0f, beta = 0.0f;
				// 真正的卷积操作 ！！！前向卷积
				checkCUDNN(hipdnnConvolutionForward(handle,
					&alpha,
					input_descriptor_mid,
					d_featuremap_1,
					kernel_descriptor_mid,
					d_kernel[i],
					convolution_descriptor_mid,
					convolution_algorithm_mid,
					d_workspace, // 注意，如果我们选择不需要额外内存的卷积算法，d_workspace可以为nullptr。
					workspace_bytes,
					&beta,
					output_descriptor_mid,
					d_featuremap_2));

				conv_bias_add << <grid, block >> > (d_featuremap_2, d_bias[i], nx, ny);


				// 前向 Relu 激活函数
				checkCUDNN(hipdnnActivationForward(handle,
					activation_descriptor,
					&alpha,
					output_descriptor_mid,
					d_featuremap_2,
					&beta,
					output_descriptor_mid,
					d_featuremap_2));
			}
			else
			{
				const float alpha = 1.0f, beta = 0.0f;
				// 真正的卷积操作 ！！！前向卷积
				checkCUDNN(hipdnnConvolutionForward(handle,
					&alpha,
					input_descriptor_mid,
					d_featuremap_2,
					kernel_descriptor_mid,
					d_kernel[i],
					convolution_descriptor_mid,
					convolution_algorithm_mid,
					d_workspace, // 注意，如果我们选择不需要额外内存的卷积算法，d_workspace可以为nullptr。
					workspace_bytes,
					&beta,
					output_descriptor_mid,
					d_featuremap_1));

				conv_bias_add << <grid, block >> > (d_featuremap_1, d_bias[i], nx, ny);


				// 前向 Relu 激活函数
				checkCUDNN(hipdnnActivationForward(handle,
					activation_descriptor,
					&alpha,
					output_descriptor_mid,
					d_featuremap_1,
					&beta,
					output_descriptor_mid,
					d_featuremap_1));
			}
			//hipdnnDestroyActivationDescriptor(activation_descriptor);
		}

		// 真正的卷积操作 ！！！前向卷积
		checkCUDNN(hipdnnConvolutionForward(handle,
			&alpha,
			input_descriptor_last,
			d_featuremap_2,
			kernel_descriptor_last,
			d_kernel[NUM_OF_CONV_LAYERS - 1],
			convolution_descriptor_last,
			convolution_algorithm_last,
			d_workspace, // 注意，如果我们选择不需要额外内存的卷积算法，d_workspace可以为nullptr。
			workspace_bytes,
			&beta,
			output_descriptor_last,
			d_featuremap_1));

		conv_bias_add << <grid, block >> > (d_featuremap_1, d_bias[NUM_OF_CONV_LAYERS - 1], nx, ny);

		// 前向 Relu 激活函数
		checkCUDNN(hipdnnActivationForward(handle,
			activation_descriptor,
			&alpha,
			output_descriptor_last,
			d_featuremap_1,
			&beta,
			output_descriptor_last,
			d_featuremap_1));

		

		if(item!=0)
			pixelShuffle_thread.join();
		hipMemcpy(outputBuffer, d_featuremap_1, buffer_size * scale_factor * scale_factor, hipMemcpyDeviceToHost);
		pixelShuffle_thread = thread(pixelShuffle, image.rows, image.cols, image_H, item, outputBuffer);
		end_t = clock();
		cout << "time: " << (double)(end_t - start_t) / CLOCKS_PER_SEC * 1000 << " ms" << endl;

		
	}
	pixelShuffle_thread.join();
	//hipFree(d_kernel);
	//hipFree(d_bias);
	//hipFree(d_featuremap_1);
	//hipFree(d_featuremap_2);
	//hipFree(d_workspace);
		//hipdnnDestroyTensorDescriptor(input_descriptor);
		//hipdnnDestroyTensorDescriptor(output_descriptor);
		//hipdnnDestroyFilterDescriptor(kernel_descriptor);
		//hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
		//hipdnnDestroy(handle);
	return 0;
}
